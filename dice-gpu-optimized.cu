#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdint.h>
#include <stdlib.h>
#include <hipcub/hipcub.hpp>
#include <inttypes.h>



__global__ void find_magnitudes(char* clks, int32_t* output){
    int index = threadIdx.x;
    int block_index = blockIdx.x;
    int32_t* clk_ints = (int32_t*) clks;

    int32_t a = clk_ints[index + 128 * block_index];
    int local_sum = 0;
    while(a){
        a = (a & (a - 1));
        local_sum++;
    }

    typedef hipcub::BlockReduce<int, 128> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    __syncthreads();
    int aggregate = BlockReduce(temp_storage).Sum(local_sum);
    if(index == 0){
        output[block_index] = aggregate;
    }
}

__global__ void find_dice_coeff(char* clks1, char* clks2, int32_t num_clks1, int_fast64_t num_clks2, int_fast64_t x_stride, int_fast64_t y_stride, int32_t* mags1, int32_t* mags2, int_fast64_t* output, float threshhold){
    typedef hipcub::BlockReduce<int, 128> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    int32_t* clk_ints1 = (int32_t*) clks1;
    int32_t* clk_ints2 = (int32_t*) clks2;

    for(int_fast64_t i = blockIdx.x; i < num_clks1; i += x_stride){
        for(int_fast64_t j = blockIdx.y; j < num_clks2; j += y_stride){
            int index = threadIdx.x;
            int_fast64_t block_index_x = i;
            int_fast64_t block_index_y = j;

            
            int32_t a = clk_ints1[index + 128 * block_index_x];
            int32_t b = clk_ints2[index + 128 * block_index_y];
            int32_t c = a & b;
            int local_sum = 0;
            while(c){
                c = (c & (c - 1));
                local_sum++;
            }

            __syncthreads();
            int dot_prod = BlockReduce(temp_storage).Sum(local_sum);
            if(index == 0){
                if(2 * dot_prod >= threshhold * (mags1[block_index_x] + mags2[block_index_y])){
                    output[block_index_x] = block_index_y + 1;
                }
            }
        }
    }
}


int main(int argc, char* argv[]){

    float threshold = 0.9;
    char* output_path = "./matches.csv";
    if (argc >= 2){
        threshold = strtod(argv[1], NULL);
    }
    if(argc >= 3){
        output_path = argv[2];
    }


    clock_t begin = clock();
    //Dataset1
    FILE* dataset1 = fopen("./dataset1.bin", "rb");
    fseek(dataset1, 0, SEEK_END);
    int_fast64_t dataset1_size_bytes = ftell(dataset1);
    rewind(dataset1);
    int_fast64_t num_clks_dataset1 = dataset1_size_bytes / 512;
    
    char* clks_dataset1 = (char*) malloc(512 * num_clks_dataset1);
    char* d_clks_dataset1;
    size_t pitch;
    hipMallocPitch(&d_clks_dataset1, &pitch, 512, num_clks_dataset1);

    int32_t* d_mags1;
    hipMalloc(&d_mags1, num_clks_dataset1 * sizeof(int32_t));

    fread(clks_dataset1, 1, 512 * num_clks_dataset1, dataset1);
    hipMemcpy2D(d_clks_dataset1, pitch, clks_dataset1, 512, 512, num_clks_dataset1, hipMemcpyHostToDevice);
    free(clks_dataset1);

    find_magnitudes<<<num_clks_dataset1, 128>>>(d_clks_dataset1, d_mags1);
    printf("Found Dataset 1 magnitudes...\n");



    //Dataset2
    FILE* dataset2 = fopen("./dataset2.bin", "rb");
    fseek(dataset2, 0, SEEK_END);
    int_fast64_t dataset2_size_bytes = ftell(dataset2);
    rewind(dataset2);
    int_fast64_t num_clks_dataset2 = dataset2_size_bytes / 512;
    
    char* clks_dataset2 = (char*) malloc(512 * num_clks_dataset2);
    char* d_clks_dataset2;
    hipMallocPitch(&d_clks_dataset2, &pitch, 512, num_clks_dataset2);

    int32_t* d_mags2;
    hipMalloc(&d_mags2, num_clks_dataset2 * sizeof(int32_t));

    fread(clks_dataset2, 1, 512 * num_clks_dataset2, dataset2);
    hipMemcpy2D(d_clks_dataset2, pitch, clks_dataset2, 512, 512, num_clks_dataset2, hipMemcpyHostToDevice);
    free(clks_dataset2);


    find_magnitudes<<<num_clks_dataset2, 128>>>(d_clks_dataset2, d_mags2);
    printf("Found Dataset 2 magnitudes...\n");
    
    //dotproducts
    int_fast64_t* d_output;
    hipMalloc(&d_output, num_clks_dataset1 * sizeof(int_fast64_t));


    dim3 threads_per_block(128);
    int_fast64_t blocks_x = 128;
    int_fast64_t blocks_y = 128;
    dim3 num_blocks(blocks_x, blocks_y);
    find_dice_coeff<<<num_blocks, threads_per_block>>>(d_clks_dataset1, d_clks_dataset2, num_clks_dataset1, num_clks_dataset2, blocks_x, blocks_y, d_mags1, d_mags2, d_output, threshold);

    int_fast64_t* output = (int_fast64_t*) malloc(num_clks_dataset1 * sizeof(int_fast64_t));
    hipMemcpy(output, d_output, num_clks_dataset1 * sizeof(int_fast64_t), hipMemcpyDeviceToHost);
    printf("Finished computation...\n");
    FILE* outfile = fopen(output_path, "w");
    for(int_fast64_t i = 0; i < num_clks_dataset1; i++){
        fprintf(outfile, "%" PRId64 ",%" PRId64 "\n", i, output[i] - 1);
    }
    printf("done.\n");
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("The elapsed time is %f seconds\n", time_spent);
    return 0;
}